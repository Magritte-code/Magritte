
#include <hip/hip_runtime.h>
#include <iostream>
using std::cout;
using std::endl;


struct array_gpu
{
    size_t  size;
    double* data;


    void* operator new (size_t len)
    {
        void* ptr;
        hipMallocManaged (&ptr, len);
        hipDeviceSynchronize ();
        return ptr;
    }

    void operator delete (void *ptr)
    {
        hipDeviceSynchronize ();
        hipFree (ptr);
    }

    array_gpu (const size_t s)
    {
//        array_gpu* this_ptr;
//        cudaMalloc (&this_ptr, sizeof(array_gpu));
//        this = this_ptr;
        size = s;
        hipMallocManaged (&data, size*sizeof(double));
        hipDeviceSynchronize ();
    }

    ~array_gpu()
    {
        hipDeviceSynchronize ();
        hipFree (data);
    }

};

struct array_cpu
{
    size_t  size;
    double* data;

    array_cpu (const size_t s)
    {
        size = s;
        data = (double*) std::malloc (size*sizeof(double));
    }

    ~array_cpu ()
    {
        std::free (data);
    }
};

void copy (array_cpu& arr_cpu, array_gpu& arr_gpu)
{
    hipMemcpy(arr_gpu.data,
               arr_cpu.data,
               arr_cpu.size*sizeof(double),
               hipMemcpyHostToDevice      );
    hipDeviceSynchronize ();
}

void copy (array_gpu& arr_gpu, array_cpu& arr_cpu)
{
    hipDeviceSynchronize ();
    hipMemcpy(arr_cpu.data,
               arr_gpu.data,
               arr_gpu.size*sizeof(double),
               hipMemcpyDeviceToHost      );
}


__global__ void kernel (array_gpu& arr_gpu)
{
    for (size_t i = 0; i < arr_gpu.size; i++)
    {
        arr_gpu.data[i]++;
    }
}




//int main ()
//{
//    const size_t size = 5;
//    double* arr = (double*) malloc(size*sizeof(double));
//
//    array a = array(size);
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr[i] = i + 4.0;
//        a.data[i] = arr[i];
//        cout << arr[i] << endl;
//    }
//
//    double* arr_dev;
//    cudaMalloc(&arr_dev, size*sizeof(double));
//
//
//    cudaMemcpy(arr_dev, arr, size*sizeof(double), cudaMemcpyHostToDevice);
//
//
//    kernel<<<1,1>>>(arr_dev);
//    cudaDeviceSynchronize();
//
//    cudaMemcpy(arr, arr_dev, size*sizeof(double), cudaMemcpyDeviceToHost);
//
//    for (size_t i = 0; i < size; i++)
//    {
//       cout << arr[i] << endl;
//    }
//
//
//    free    (arr);
//    cudaFree(arr_dev);
//
//    return (0);
//}


int main ()
{
    const size_t size = 5;

    array_cpu* arr_cpu_ptr = new array_cpu (size);
    array_gpu* arr_gpu_ptr = new array_gpu (size);

    array_cpu arr_cpu = *arr_cpu_ptr;
    array_gpu arr_gpu = *arr_gpu_ptr;

    for (size_t i = 0; i < size; i++)
    {
        arr_cpu.data[i] = i + 4.0;
        cout << arr_cpu.data[i] << endl;
    }

    copy (arr_cpu, arr_gpu);

    cout << "on gpu ---" << endl;
    cout << &arr_gpu     << endl;
    cout <<  arr_gpu_ptr << endl;

    cout << "on cpu ---" << endl;
    cout << &arr_cpu     << endl;
    cout <<  arr_cpu_ptr << endl;

    kernel<<<1,1>>>(*arr_gpu_ptr);
    hipDeviceSynchronize();

    copy (arr_gpu, arr_cpu);

    for (size_t i = 0; i < size; i++)
    {
        cout << "--- " << arr_cpu.data[i] << endl;
    }

    return (0);
}
