#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::endl;
#include <vector>
using std::vector;

#include "paracabs.hpp"
using namespace paracabs::accelerator;
using paracabs::datatypes::vector3d;
using paracabs::datatypes::array1d;
using paracabs::datatypes::MemTypeDefault;
using paracabs::datatypes::MemTypeAccelerator;
using paracabs::datatypes::my_copy;


typedef vector3d<double, MemTypeDefault>     Vector3d;
typedef vector3d<double, MemTypeAccelerator> Vector3d_accel;

typedef array1d <Vector3d,       MemTypeDefault>     arrvec3d;
typedef array1d <Vector3d_accel, MemTypeAccelerator> arrvec3d_accel;

typedef array1d <double, MemTypeDefault>     arrdouble;
typedef array1d <double, MemTypeAccelerator> arrdouble_accel;


__global__ void printKernel (arrdouble_accel* arr)
{
    for (size_t i = 0; i < arr->size; i++)
    {
        printf("test");
    }
}


int main ()
{
    cout << "Paracabs test accelerator."   << endl;
    cout << "Number of GPUs = " << nGPUs() << endl;


    list_accelerators();

    const size_t size = 10;

//    arrvec3d*       arr_cpu = new arrvec3d       (size);
//    arrvec3d_accel* arr_gpu = new arrvec3d_accel (size);

    arrdouble*       arr_cpu = new arrdouble       (size);
//    arrdouble_accel* arr_gpu = new arrdouble_accel (size);


    for (int i = 0; i < size; i++)
    {
        arr_cpu[i] = i + 4;
    }

//    paracabs::datatypes::my_copy <double> (arr_cpu, arr_gpu);

//    printKernel <<<1, 1>>> (arr_gpu);
//    paracabs::accelerator::synchronize();

    delete arr_cpu;
//    delete arr_gpu;

    cout << "Done." << endl;

    return (0);
}