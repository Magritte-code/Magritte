#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::endl;
#include <vector>
using std::vector;

#include "paracabs.hpp"
using namespace paracabs::accelerator;
//using paracabs::datatypes::vector3d;
//using paracabs::datatypes::array1d;
using paracabs::datatypes::MemTypeDefault;
using paracabs::datatypes::MemTypeAccelerator;
using paracabs::datatypes::my_copy;

//template <typename T>
//typedef  vector<T, > Vector<T>;

//typedef vector3d<double, MemTypeDefault>     Vector3d;
//typedef vector3d<double, MemTypeAccelerator> Vector3d_accel;

//typedef array1d <Vector3d,       MemTypeDefault>     arrvec3d;
//typedef array1d <Vector3d_accel, MemTypeAccelerator> arrvec3d_accel;

//typedef array1d <double, MemTypeDefault>     arrdouble;
//typedef array1d <double, MemTypeAccelerator> arrdouble_accel;


__global__ void printKernel (vector <double, paracabs::allocator<double, MemTypeAccelerator>> vec)
{
    for (size_t i = 0; i < vec.size(); i++)
    {
        printf("test\n");
    }
}


int main ()
{
    cout << "Paracabs test allocator."     << endl;
    cout << "Number of GPUs = " << nGPUs() << endl;

    list_accelerators();

    const size_t size = 10;

    vector <double, paracabs::allocator<double, MemTypeDefault>>     vec       (10);
    vector <double, paracabs::allocator<double, MemTypeAccelerator>> vec_accel (10);

    my_copy (vec, vec_accel);

    cout << "Done." << endl;

    return (0);
}