#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::endl;
#include <vector>
using std::vector;

#include "paracabs.hpp"
namespace pc = paracabs::datatypes;

using Vector3D = pc::Vector3D<double, pc::MemTypeDefault>;

template <typename type>
using Array     = pc::Array <type, pc::MemTypeDefault>;
template <typename type>
using Array_acc = pc::Array <type, pc::MemTypeAccelerator>;


struct test
{
    Array     <double>*   nums;
    Array_acc <double>*   nums_acc;
    Array     <Vector3D>* vecs;
    Array_acc <Vector3D>* vecs_acc;


};


__global__ void addKernel ()
{

}


int main ()
{
    cout << "Paracabs test datatypes." << endl;

    Vector3D v1 (1.0, 2.0, 3.0);
    Vector3D v2 (4.0, 5.0, 6.0);
    Vector3D v3 = v1 + v2;

    v1.print();
    v2.print();
    v3.print();

    cout << v1.dot(v2) << endl;


    v1 += v2;

    v1.print();
    v2.print();


    Vector3D v4 = 3.14;

    v4.print();

    (v4 + 1).print();

    v4.print();

    v4 = 7.12;

    v4.print();


    const size_t size = 10;

//    array1d <vector3d <double>, MemTypeDefault>     arr1 (size);
//    array1d <vector3d <double>, MemTypeAccelerator> arr2 (size);
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr1[i] = 1.0;
//    }
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr1[i].print();
//    }
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr2[i] = 1.0;
//    }
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr2[i].print();
//    }


    cout << "Done." << endl;

    return (0);
}